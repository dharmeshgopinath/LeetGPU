#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {

int i = blockIdx.x * blockDim.x + threadIdx.x;

if(i<N){
    C[i] = A[i] + B[i];
}
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int N) {
    
    //Mem Alloacation in GPU -> VRAM
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc((void**) &d_A, N*sizeof(float));
    hipMalloc((void**) &d_B, N*sizeof(float));
    hipMalloc((void**) &d_C, N*sizeof(float));

    
    hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);


    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize(); //used to wait for GPU to finish all processes

    // Free memory in device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
